#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

__global__ void square_kernel (float* x, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = x[idx] * x[idx];
    }
}

void launch_square (torch::Tensor input, torch::Tensor output) {
    int size    = input.numel ();
    int threads = 256;
    int blocks  = (size + threads - 1) / threads;

    square_kernel<<<blocks, threads>>> (
    input.data_ptr<float> (), output.data_ptr<float> (), size);
}
